#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <iostream>
#include <numeric>
#include <map>
#include <sys/time.h>
#include <valarray>

#include <hdf5.h>

#include "range.hpp"
#include "utils.hpp"

#define NUM_ROWS 28
#define NUM_COLS 28
#define NUM_CHANNELS 1
#define NUM_DIGITS 10

#define TILE_WIDTH 16

static int FLAGS_batch_size = 10000;
static std::string FLAGS_testdata{};
static std::string FLAGS_model{};

// Data and reference data dimensions
static int xdims[] = {FLAGS_batch_size, NUM_ROWS, NUM_COLS, NUM_CHANNELS};
static int rdims[] = {FLAGS_batch_size, NUM_DIGITS};

// Model dimensions
static int conv1dims[] = {5, 5, 1, 32};
static int conv2dims[] = {5, 5, 32, 64};
static int fc1dims[]   = {1024, 128};
static int fc2dims[]   = {128, 10};

static int loadData(float *x, float *y) {
    // Open the data file
    const auto file_id =
        H5Fopen(FLAGS_testdata.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

    // Open the dataset x and y
    const auto x_id = H5Dopen2(file_id, "/x", H5P_DEFAULT);
    const auto y_id = H5Dopen2(file_id, "/y", H5P_DEFAULT);

    // Get the dataset x dimensions
    const auto xspace = H5Dget_space(x_id);
    const auto xndims = H5Sget_simple_extent_ndims(xspace);
    assert(xndims == 4);

    hsize_t input_dims[xndims];
    H5Sget_simple_extent_dims(xspace, input_dims, NULL);
    if (input_dims[0] != FLAGS_batch_size) {
        std::cout << "data size does not match batch size specified!\n";
        return 1; // return error
    }
    std::cout << "input dimensions = " << input_dims[0] << " x " << input_dims[1]
              << " x " << input_dims[2] << " x " << input_dims[3] << "\n";

    // Read the dataset x and y
    check_success(
        H5Dread(x_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, x));
    check_success(
        H5Dread(y_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, y));

    // Close the dataset x and y
    check_success(H5Dclose(x_id));
    check_success(H5Dclose(y_id));

    // Close the file
    check_success(H5Fclose(file_id));

    // return success
    return 0;
}

static void loadModel(float *conv1, float *conv2, float *fc1, float *fc2) {
    // Open the model file
    const auto file_id = H5Fopen(FLAGS_model.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

    // Open the dataset
    const auto conv1_id = H5Dopen2(file_id, "/conv1", H5P_DEFAULT);
    const auto conv2_id = H5Dopen2(file_id, "/conv2", H5P_DEFAULT);
    const auto fc1_id   = H5Dopen2(file_id, "/fc1", H5P_DEFAULT);
    const auto fc2_id   = H5Dopen2(file_id, "/fc2", H5P_DEFAULT);

    // Read the dataset
    check_success(H5Dread(conv1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                          H5P_DEFAULT, conv1));
    check_success(H5Dread(conv2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                          H5P_DEFAULT, conv2));
    check_success(
        H5Dread(fc1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc1));
    check_success(
        H5Dread(fc2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc2));

    // Close the dataset x and y
    check_success(H5Dclose(conv1_id));
    check_success(H5Dclose(conv2_id));
    check_success(H5Dclose(fc1_id));
    check_success(H5Dclose(fc2_id));

    // Close the file
    check_success(H5Fclose(file_id));
}

// CUDA kernel for forward convolution path

/* __global__ void conv_forward_kernel(float *X, float *W, float *Y, int xdims[4], int wdims[4], int ydims[4])
 * Local Variables:
 *      n = sample index
 *      m = output map index
 *      w = column index within an output map
 *      h = row index within an output map
 *      p = row index within a mask
 *      q = column index within a mask
 *      c = input map/channel index
 */
__global__ void conv_forward_kernel(float *X, float *W, float *Y, int xdims[4], int wdims[4], int ydims[4])
{
    int n, m, h0, w0, h_base, w_base, h, w;

    int X_tile_width = TILE_WIDTH + wdims[0]-1;
    extern __shared__ float shmem[];
    float* X_shared = &shmem[0];
    float* W_shared = &shmem[X_tile_width * X_tile_width];

    n = blockIdx.x;
    m = blockIdx.y;

    h0 = threadIdx.y;
    w0 = threadIdx.x;
    h_base = (blockIdx.z / ((ydims[2]-1)/TILE_WIDTH + 1))*TILE_WIDTH;
    w_base = (blockIdx.z % ((ydims[1]-1)/TILE_WIDTH + 1))*TILE_WIDTH;
    h = h_base + h0;
    w = w_base + w0;

    float acc = 0.0f;

    // sum over all input channels
    for (int c = 0; c < wdims[2]  ; c++)
    {
        // load weights for W [m, c,..], h0 and w0 used as shorthand for threadIdx.x and threadIdx.y
        int woffset = h0*wdims[1]*wdims[2]*wdims[3] + w0*wdims[2]*wdims[3] + c*wdims[3] + m;
        if (( h0 < wdims[0]) && ( w0 < wdims[1])) W_shared[h0*wdims[1] + w0] = W[woffset];

        __syncthreads();

        // load tile from X[n, c,...] into shared memory
        for (int i = h; i < h_base + X_tile_width; i += TILE_WIDTH)
        {
            for (int j = w; j < w_base + X_tile_width; j += TILE_WIDTH)
            {
                int xoffset = n*xdims[1]*xdims[2]*xdims[3] + i*xdims[2]*xdims[3] + j*xdims[3] + c;
                int x_shared_offset = X_tile_width*(i - h_base) + (j - w_base);
                X_shared[x_shared_offset] = X[xoffset];
            }
        }

        __syncthreads();

        for (int p = 0; p < wdims[0]; p++)
        {
            for (int q = 0; q < wdims[1]; q++)
            {
                int x_shared_offset = X_tile_width*(h0+p) + (w0+q);
                int w_shared_offset = wdims[1]*p + q;
                acc = acc + X_shared[x_shared_offset] * W_shared[w_shared_offset];
            }
        }

        __syncthreads();
    }

    int yoffset = ((n * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;

    if(w < ydims[2] && h < ydims[1])
        Y[yoffset] = (acc < 0.0f) ? 0.0f : acc;
}

// CUDA kernel for average pool
__global__ void average_pool_kernel(float *X, float *Y, int xdims[4], int ydims[4], int pool_size) {

    int n, m, h, w;
    n = blockIdx.x;
    m = blockIdx.y;

    h = (blockIdx.z / ((ydims[2]-1)/TILE_WIDTH + 1))*TILE_WIDTH + threadIdx.y;
    w = (blockIdx.z % ((ydims[1]-1)/TILE_WIDTH + 1))*TILE_WIDTH + threadIdx.x;

    int i = n;

    if(w < ydims[2] && h < ydims[1]){
        float acc = 0.0f;
        for (int p = 0; p < pool_size; p++) {
            for (int q = 0; q < pool_size; q++) {
                int xoffset = i * xdims[1] * xdims[2] * xdims[3] + (pool_size * h + p) * xdims[2] * xdims[3] + (pool_size * w + q) * xdims[3] + m;
                acc += X[xoffset] / (1.0f * pool_size * pool_size);
            }
        }
        int yoffset = ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
        Y[yoffset] = acc;
    }
}

static void fully_forward(const float *X, const int xdims[2], float *W,

                          const int wdims[2], float *Y, const int ydims[2]) {
    for (const auto i : range(0, xdims[0])) {
        for (const auto j : range(0, wdims[1])) {
            float sum = 0;
            for (const auto k : range(0, xdims[1])) {
                sum += X[i * xdims[1] + k] * W[k * wdims[1] + j];
            }
            Y[i * wdims[1] + j] = (sum < 0.0f) ? 0.0f : sum;
        }
    }
}

// Choose the guess with largest score
static void argmax(const float *X, const int xdims[2], int *Y) {
    for (const auto i : range(0, xdims[0])) {
        auto max_idx = 0;
        auto max     = X[i * xdims[1]];
        for (const auto j : range(0, xdims[1])) {
            const auto elem = X[(i * xdims[1]) + j];
            if (elem > max) {
                max_idx = j;
                max     = elem;
            }
        }
        Y[i] = max_idx;
    }
}

// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {

    // conv layer 1 vars
    const int adims[] = {xdims[0], (xdims[1] - conv1dims[0] + 1),
                         (xdims[2] - conv1dims[1] + 1), conv1dims[3]};
    auto a = zeros<float>(adims);

    // avg pool 1 vars
    const int pool_size = 2;
    const int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size,
                           adims[3]};
    auto b = zeros<float>(bdims);

    // conv layer 2 vars
    const int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
                         (bdims[2] - conv2dims[1] + 1), conv2dims[3]};
    auto c = zeros<float>(cdims);

    // avg pool 2 vars
    const int ddims[] = {cdims[0], cdims[1] / pool_size, cdims[2] / pool_size,
                         cdims[3]};
    auto d = zeros<float>(ddims);
    auto pool2Output = zeros<float>(ddims);

    // fully connected layer 1 vars
    const int ddims2[] = {ddims[0], ddims[1] * ddims[2] * ddims[3]};
    const int edims[] = {ddims[0], fc1dims[1]};
    auto e = zeros<float>(edims);

    // fully connected layer 2 vars
    const int fdims[] = {edims[0], fc2dims[1]};
    auto f = zeros<float>(fdims);

    // CUDA device vars
    int * deviceIndims, * deviceMaskdims, * deviceOutdims;               // logistical vars
    float * deviceInputConv1, * deviceMaskConv1, * deviceOutputConv1;    // conv 1 vars
    float * deviceInputPool1, * deviceOutputPool1;                       // pool 1 vars
    float * deviceInputConv2, * deviceMaskConv2, * deviceOutputConv2;    // conv 2 vars
    float * deviceInputPool2, * deviceOutputPool2;                       // pool 2 vars
    float * deviceInputFullyForward1, * deviceOutputFullyForward1;       // fully connected 1 vars
    float * deviceInputFullyForward2, * deviceOutputFullyForward2;       // fully connected 2 vars

    fully_forward(pool2Output, ddims2, fc1, fc1dims, e, edims);

    // allocate memory for device data dims
    check_success(hipMalloc((void**)&deviceIndims, 4*sizeof(int)));
    check_success(hipMalloc((void**)&deviceMaskdims, 4*sizeof(int)));
    check_success(hipMalloc((void**)&deviceOutdims, 4*sizeof(int)));

    /*********************************************** CONV 1 Layer ************************************************/
    // allocate memory for device data
    check_success(hipMalloc((void**)&deviceInputConv1, xdims[0]*xdims[1]*xdims[2]*conv1dims[2]*xdims[3]*sizeof(float)));
    check_success(hipMalloc((void**)&deviceMaskConv1, conv1dims[0]*conv1dims[1]*conv1dims[2]*conv1dims[3]*xdims[3]*sizeof(float)));
    check_success(hipMalloc((void**)&deviceOutputConv1, adims[0]*adims[1]*adims[2]*adims[3]*xdims[3]*sizeof(float)));

    // copy data to device
    check_success(hipMemcpy(deviceInputConv1, x, xdims[0]*xdims[1]*xdims[2]*conv1dims[2]*xdims[3]*sizeof(float),hipMemcpyHostToDevice));
    check_success(hipMemcpy(deviceMaskConv1, conv1, conv1dims[0]*conv1dims[1]*conv1dims[2]*conv1dims[3]*xdims[3]*sizeof(float),hipMemcpyHostToDevice));
    // copy data dims to device
    check_success(hipMemcpy(deviceIndims, xdims, 4*sizeof(int),hipMemcpyHostToDevice));
    check_success(hipMemcpy(deviceMaskdims, conv1dims, 4*sizeof(int),hipMemcpyHostToDevice));
    check_success(hipMemcpy(deviceOutdims, adims, 4*sizeof(int),hipMemcpyHostToDevice));

    // kernel dims
    int N = xdims[0];
    int M = conv1dims[3];
    int Z = ((adims[2]-1)/TILE_WIDTH+1)*((adims[1]-1)/TILE_WIDTH+1);
    dim3 blockDimConv1(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDimConv1(N, M, Z);

    unsigned int shmem_size_1 = sizeof(float)*((TILE_WIDTH + conv1dims[1] - 1)*(TILE_WIDTH + conv1dims[0] - 1) + conv1dims[0]*conv1dims[1]);

    // first kernel launch
    conv_forward_kernel<<<gridDimConv1, blockDimConv1, shmem_size_1>>>(deviceInputConv1, deviceMaskConv1, deviceOutputConv1, deviceIndims, deviceMaskdims, deviceOutdims);
    hipDeviceSynchronize();

    // simply use the device output data as input for the next kernel launch
    deviceInputPool1 = deviceOutputConv1;

    // Free memory for conv1
    hipFree(deviceInputConv1);
    hipFree(deviceMaskConv1);

    /*********************************************** AVG POOL 1 Layer ************************************************/
    // allocate memory for device pool 1 calculation
    check_success(hipMalloc((void**)&deviceOutputPool1, bdims[0]*bdims[1]*bdims[2]*bdims[3]*xdims[3]*sizeof(float)));

    // copy data dims to device
    check_success(hipMemcpy(deviceIndims, adims, 4*sizeof(int),hipMemcpyHostToDevice));
    check_success(hipMemcpy(deviceOutdims, bdims, 4*sizeof(int),hipMemcpyHostToDevice));

    // kernel dims
    N = adims[0];
    M = conv1dims[3];
    Z = ((bdims[2]-1)/TILE_WIDTH+1)*((bdims[1]-1)/TILE_WIDTH+1);//adims[2]*adims[1];
    dim3 blockDimPool1(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDimPool1(N, M, Z);

    // avg pool 1 launch
    average_pool_kernel<<<gridDimPool1, blockDimPool1>>>(deviceInputPool1, deviceOutputPool1, deviceIndims, deviceOutdims, pool_size);
    hipDeviceSynchronize();

    // simply use the device output data as input for the next kernel launch
    deviceInputConv2 = deviceOutputPool1;

    // avg pool memory freed
    hipFree(deviceInputPool1);

    /*********************************************** CONV 2 Layer ************************************************/
    // conv layer 2 setup
    check_success(hipMalloc((void**)&deviceMaskConv2, conv2dims[0]*conv2dims[1]*conv2dims[2]*conv2dims[3]*xdims[3]*sizeof(float)));
    check_success(hipMalloc((void**)&deviceOutputConv2, cdims[0]*cdims[1]*cdims[2]*cdims[3]*xdims[3]*sizeof(float)));

    // copy data to device
    check_success(hipMemcpy(deviceMaskConv2, conv2, conv2dims[0]*conv2dims[1]*conv2dims[2]*conv2dims[3]*xdims[3]*sizeof(float),hipMemcpyHostToDevice));

    // copy data dims to device
    check_success(hipMemcpy(deviceIndims, bdims, 4*sizeof(int),hipMemcpyHostToDevice));
    check_success(hipMemcpy(deviceMaskdims, conv2dims, 4*sizeof(int),hipMemcpyHostToDevice));
    check_success(hipMemcpy(deviceOutdims, cdims, 4*sizeof(int),hipMemcpyHostToDevice));

    // kernel dims
    N = bdims[0];
    M = conv2dims[3];
    Z = ((cdims[2]-1)/TILE_WIDTH+1)*((cdims[1]-1)/TILE_WIDTH+1);
    dim3 blockDimConv2(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDimConv2(N, M, Z);

    unsigned int shmem_size_2 = sizeof(float)*((TILE_WIDTH + conv2dims[1] - 1)*(TILE_WIDTH + conv2dims[0] - 1) + conv2dims[0]*conv2dims[1]);

    // conv layer 2
    conv_forward_kernel<<<gridDimConv2, blockDimConv2, shmem_size_2>>>(deviceInputConv2, deviceMaskConv2, deviceOutputConv2, deviceIndims, deviceMaskdims, deviceOutdims);
    hipDeviceSynchronize();

    // simply use the device output data as input for the next kernel launch
    deviceInputPool2 = deviceOutputConv2;

    // freeing device memory for conv 2 layer
    hipFree(deviceInputConv2);
    hipFree(deviceMaskConv2);

    /*********************************************** AVG POOL 2 Layer ************************************************/
    // allocate memory for device pool 2 calculation
    check_success(hipMalloc((void**)&deviceOutputPool2, ddims[0]*ddims[1]*ddims[2]*ddims[3]*xdims[3]*sizeof(float)));

    // copy data dims to device
    check_success(hipMemcpy(deviceIndims, cdims, 4*sizeof(int),hipMemcpyHostToDevice));
    check_success(hipMemcpy(deviceOutdims, ddims, 4*sizeof(int),hipMemcpyHostToDevice));

    // kernel dims
    N = cdims[0];
    M = conv2dims[3];
    Z = ((ddims[2]-1)/TILE_WIDTH+1)*((ddims[1]-1)/TILE_WIDTH+1);//adims[2]*adims[1];
    dim3 blockDimPool2(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDimPool2(N, M, Z);

    // Second average pool kernel launch
    average_pool_kernel<<<gridDimPool2, blockDimPool2>>>(deviceInputPool2, deviceOutputPool2, deviceIndims, deviceOutdims, pool_size);
    hipDeviceSynchronize();

    // copy output data back from device
    check_success(hipMemcpy(pool2Output, deviceOutputPool2, ddims[0]*ddims[1]*ddims[2]*ddims[3]*xdims[3]*sizeof(float), hipMemcpyDeviceToHost));

    // avg pool memory freed
    hipFree(deviceInputPool2);
    hipFree(deviceOutputPool2);

    /*********************************************** FULLY CONNECTED 1 Layer ************************************************/
    fully_forward(pool2Output, ddims2, fc1, fc1dims, e, edims);

    /*********************************************** FULLY CONNECTED 2 Layer ************************************************/
    fully_forward(e, edims, fc2, fc2dims, f, fdims);

    /*********************************************** GAUSSIAN Layer ************************************************/
    argmax(f, fdims, out);

    // freeing host buffers
    delete[] a;
    delete[] b;
    delete[] c;
    delete[] d;
    delete[] e;
    delete[] f;
    delete[] pool2Output;

    // freeing device memory for dimensional data
    hipFree(deviceIndims);
    hipFree(deviceMaskdims);
    hipFree(deviceOutdims);
}

int main(int argc, char **argv) {

    if (argc != 3 && argc != 4) {
        std::cerr << "\n"
                  << "This program performs the forward opertion step for "
            "Convolutional Neural Network(CNN).  "
            "Sample usage: \n"
                  << argv[0]
                  << " [../data/test10.hdf5] [../data/model.hdf5] [10]\n";
        return -1;
    }
    FLAGS_testdata = std::string(argv[1]);
    FLAGS_model    = std::string(argv[2]);
    if (argc == 3) {
        const std::map<std::string, int> default_batch_sizes{
            {"../data/test2.hdf5", 2},
            {"../data/test10.hdf5", 10},
            {"../data/test100.hdf5", 100},
            {"../data/testfull.hdf5", 10000}};
        const auto batch_size_in_map = default_batch_sizes.find(FLAGS_testdata);
        if (batch_size_in_map == default_batch_sizes.end()) {
            std::cerr << "\nERROR:: Unrecognized file " << FLAGS_testdata << " batch_size must be specified.\n";
            return -1;
        }
        FLAGS_batch_size = batch_size_in_map->second;
    } else if (argc == 4) {
        FLAGS_batch_size = atoi(argv[3]);
    }
    xdims[0] = FLAGS_batch_size;
    rdims[0] = FLAGS_batch_size;

    // Load data into x and y
    float *x = allocate<float>(xdims);
    float *y = allocate<float>(rdims);
    loadData(x, y);

    // Load model
    float *conv1 = allocate<float>(conv1dims);
    float *conv2 = allocate<float>(conv2dims);
    float *fc1   = allocate<float>(fc1dims);
    float *fc2   = allocate<float>(fc2dims);
    loadModel(conv1, conv2, fc1, fc2);

    // Perform foward opertion
    int *out = zeros<int>(FLAGS_batch_size);

    // get start time
    const auto start = now();

    forward_operation(x, conv1, conv2, fc1, fc2, out);

    // get end time
    const auto end = now();

    // get elapsed time in milliseconds
    const auto elapsed =
        std::chrono::duration<double, std::milli>(end - start).count();

    // Get reference
    int *ref = zeros<int>(FLAGS_batch_size);
    argmax(y, rdims, ref);

    // Calculate correctness
    int num_correct = 0;
    for (const auto i : range(0, FLAGS_batch_size)) {
        if (out[i] == ref[i]) {
            num_correct++;
        }
    }
    std::cout << "Done with " << FLAGS_batch_size << " queries in "
              << "elapsed = " << elapsed << " milliseconds. Correctness: "
              << static_cast<float>(num_correct) / FLAGS_batch_size << "\n";

    delete[] x;
    delete[] y;
    delete[] conv1;
    delete[] conv2;
    delete[] fc1;
    delete[] fc2;
    delete[] out;
    delete[] ref;

    return 0;
}
